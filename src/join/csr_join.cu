#include "csr_join.h"
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include <cstdio>
#include "../relation/relation.cuh"
#include "../relation/csr_relation.h"
#include "../util.h"

CSR_Join::CSR_Join(int a, int b, int c) {
    dimA = a;
    dimB = b;
    dimC = c;
}

Relation<2> CSR_Join::join(Relation<2> rel1, Relation<2> rel2) {
    hipsparseHandle_t handle;
    CUSPARSE_CHECK(hipsparseCreate(&handle));    

    std::stringstream name;
    name << "CSR Join (" << rel1.count << ", " << rel2.count << ")";
    Timer t(name.str().c_str());

    float alpha = 1;
    float beta = 0;

    hipsparseSpMatDescr_t mat1, mat2, matOut;

    CSRMatrix rel1csr(rel1, dimA, dimB);
    CSRMatrix rel2csr(rel2, dimB, dimC);

    CUSPARSE_CHECK(hipsparseCreateCsr(&mat1, dimA, dimB, rel1.count,
            rel1csr.rowOffsets, rel1csr.columnIndexes, rel1csr.values,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateCsr(&mat2, dimB, dimC, rel2.count,
            rel2csr.rowOffsets, rel2csr.columnIndexes, rel2csr.values,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    CUSPARSE_CHECK(hipsparseCreateCsr(&matOut, dimA, dimC, 0,
            nullptr, nullptr, nullptr,
            HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
            HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));

    hipsparseSpGEMMDescr_t spgemmDesc;
    hipsparseSpGEMM_createDescr(&spgemmDesc);

    t.lap("Initialization");

    size_t bufferSize1 = 0;
    void* dBuffer1 = nullptr;

    CUSPARSE_CHECK(hipsparseSpGEMM_workEstimation(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, mat1, mat2, &beta, matOut,
        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc, &bufferSize1, nullptr));

    hipMalloc(&dBuffer1, bufferSize1);

    CUSPARSE_CHECK(hipsparseSpGEMM_workEstimation(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, mat1, mat2, &beta, matOut,
        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc, &bufferSize1, dBuffer1));

    t.lap("Work estimation");

    size_t bufferSize2 = 0;
    void* dBuffer2 = nullptr;

    CUSPARSE_CHECK(hipsparseSpGEMM_compute(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, mat1, mat2, &beta, matOut,
        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc, &bufferSize2, nullptr));

    hipMalloc(&dBuffer2, bufferSize2);

    CUSPARSE_CHECK(hipsparseSpGEMM_compute(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, mat1, mat2, &beta, matOut,
        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc, &bufferSize2, dBuffer2));

    t.lap("Core MMUL");

    // Output buffers
    int64_t outRows, outCols, outNumZero;
    hipsparseSpMatGetSize(matOut, &outRows, &outCols, &outNumZero);
    CSRMatrix outcsr;
    outcsr.numRows = outRows;
    outcsr.numNonZeros = outNumZero;

    hipMalloc(&outcsr.rowOffsets, sizeof(int) * (outcsr.numRows + 1));
    hipMalloc(&outcsr.columnIndexes, sizeof(int) * outcsr.numNonZeros);
    hipMalloc(&outcsr.values, sizeof(float) * outcsr.numNonZeros);

    CUSPARSE_CHECK(hipsparseCsrSetPointers(matOut, outcsr.rowOffsets, outcsr.columnIndexes, outcsr.values));

    CUSPARSE_CHECK(hipsparseSpGEMM_copy(
        handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, mat1, mat2, &beta, matOut,
        HIP_R_32F, HIPSPARSE_SPGEMM_DEFAULT,
        spgemmDesc));

    Relation<2> outRel = outcsr.toRelation();

    t.lap("Matrix to relation");

    // Cleanup
    CUSPARSE_CHECK(hipsparseDestroySpMat(mat1));
    CUSPARSE_CHECK(hipsparseDestroySpMat(mat2));
    CUSPARSE_CHECK(hipsparseDestroySpMat(matOut));
    CUSPARSE_CHECK(hipsparseSpGEMM_destroyDescr(spgemmDesc));
    hipFree(dBuffer1);
    hipFree(dBuffer2);

    t.finish();
    hipsparseDestroy(handle);

    return outRel;
}
