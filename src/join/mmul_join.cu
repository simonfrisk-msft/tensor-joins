#include "hip/hip_runtime.h"
#include "mmul_join.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <sstream>
#include <cstdio>
#include "../util.h"

#define MM_BLOCK 32
typedef int8_t IN_MAT;
typedef int32_t OUT_MAT;

int roundUpToBlock(int num) {
    return ((num + MM_BLOCK - 1) / MM_BLOCK) * MM_BLOCK;
}

__global__ void RelationToMatrix(Relation rel, IN_MAT* matrix, int stride) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < rel.count) {
        Tuple tuple = rel.data[idx];
        matrix[tuple.x + tuple.y * stride] = 1;
    }
}

__global__ void CountOutputSizePerBlock(OUT_MAT* matrix, int n, int m, int* outputSizePerBlock) {
    __shared__ int count;
    if (threadIdx.x == 0 && threadIdx.y == 0)
        count = 0;
    __syncthreads();
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx % n;
    int y = idx / n;
    int block = blockIdx.x;
    if(x < n && y < m && matrix[idx] > 0)
        atomicAdd(&count, 1);
    __syncthreads();
    if (threadIdx.x == 0 && threadIdx.y == 0)
        outputSizePerBlock[block] = count;
}

__global__ void MatrixToRelation(Relation out, OUT_MAT* matrix, int n, int m, int* prefixOutputSizePerBlock) {
    __shared__ int count;
    if (threadIdx.x == 0 && threadIdx.y == 0)
        count = 0;
    __syncthreads();
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int x = idx % n;
    int y = idx / n;
    int block = blockIdx.x;
    int globalOffset = prefixOutputSizePerBlock[block];
    if(x < n && y < m && matrix[idx] > 0) {
       int outIdx = atomicAdd(&count, 1);
       out.data[globalOffset + outIdx] = Tuple{ x: x, y: y };
    }
}

MMUL_Join::MMUL_Join(int a, int b, int c) {
    dimA = roundUpToBlock(a);
    dimB = roundUpToBlock(b);
    dimC = roundUpToBlock(c);
}

Relation MMUL_Join::join(Relation rel1, Relation rel2) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    std::stringstream name;
    name << "MMUL Join (" << rel1.count << ", " << rel2.count << ")";
    Timer t(name.str().c_str());

    int alpha = 1;
    int beta = 0;

    IN_MAT *M1, *M2;
    OUT_MAT *outMatrix;
    Relation outRel;

    int M1Size = dimA * dimB * sizeof(IN_MAT);
    int M2Size = dimB * dimC * sizeof(IN_MAT);
    int outSize = dimA * dimC * sizeof(OUT_MAT);
    CUDA_CHECK(hipMalloc(&M1, M1Size));
    CUDA_CHECK(hipMalloc(&M2, M2Size));
    CUDA_CHECK(hipMalloc(&outMatrix, outSize));

    int blockSizeRelToMat = 1024;
    int numBlocksM1 = (rel1.count + blockSizeRelToMat - 1) / blockSizeRelToMat;
    int numBlocksM2 = (rel2.count + blockSizeRelToMat - 1) / blockSizeRelToMat;

    if (rel1.count <= 0 || rel2.count <= 0) {
        outRel.count = 0;
        hipMalloc(&outRel.data, 0);
        return outRel;
    }

    RelationToMatrix<<<numBlocksM1, blockSizeRelToMat>>>(rel1, M1, dimA);
    RelationToMatrix<<<numBlocksM2, blockSizeRelToMat>>>(rel2, M2, dimB);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    t.lap("Relation to Matrix");

    hipblasStatus_t mmul_status = hipblasGemmEx(handle, 
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        dimA, dimC, dimB, &alpha,
        M1, HIP_R_8I, dimA,
        M2, HIP_R_8I, dimB,
        &beta,
        outMatrix, HIP_R_32I, dimC,
        HIP_R_32I,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipDeviceSynchronize();
    CUBLAS_CHECK(mmul_status);
    CUDA_CHECK(hipGetLastError());

    t.lap("Matrix Multiplication");

    int relToMatrixBlock = 1024;
    int blockCountRelToMatrix = (dimA*dimC + relToMatrixBlock - 1) / relToMatrixBlock;

    int* outputSizePerBlock;
    int* prefixOutputSizePerBlock;
    CUDA_CHECK(hipMalloc(&outputSizePerBlock, blockCountRelToMatrix * sizeof(int)));
    CUDA_CHECK(hipMalloc(&prefixOutputSizePerBlock, (blockCountRelToMatrix+1) * sizeof(int)));
    CUDA_CHECK(hipMemset(prefixOutputSizePerBlock, 0, sizeof(int))); // set first offset to 0

    CountOutputSizePerBlock<<<blockCountRelToMatrix, relToMatrixBlock>>>(outMatrix, dimA, dimC, outputSizePerBlock);
    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());
    t.lap("Output counting");

    thrust::device_ptr<int> thrust_ptr(outputSizePerBlock);
    thrust::device_ptr<int> thrust_prefix_ptr(prefixOutputSizePerBlock);
    thrust::inclusive_scan(thrust_ptr, thrust_ptr + blockCountRelToMatrix, thrust_prefix_ptr+1);
    CUDA_CHECK(hipMemcpy(&outRel.count, prefixOutputSizePerBlock + blockCountRelToMatrix, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMalloc(&outRel.data, outRel.count * sizeof(Tuple)));
    t.lap("Prefix sum");

    MatrixToRelation<<<blockCountRelToMatrix, relToMatrixBlock>>>(outRel, outMatrix, dimA, dimC, prefixOutputSizePerBlock);
    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());
    t.lap("Matrix to Relation");

    CUDA_CHECK(hipFree(outMatrix));
    CUDA_CHECK(hipFree(M1));
    CUDA_CHECK(hipFree(M2));
    
    t.finish();
    hipblasDestroy(handle);

    return outRel;
}
