#include "mmul_join.h"
#include <hipblas.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include <sstream>
#include <cstdio>
#include "../relation.h"
#include "../util.h"

#define MM_BLOCK 32

int roundUpToBlock(int num) {
    return ((num + MM_BLOCK - 1) / MM_BLOCK) * MM_BLOCK;
}

MMUL_Join::MMUL_Join(int a, int b, int c) {
    dimA = roundUpToBlock(a);
    dimB = roundUpToBlock(b);
    dimC = roundUpToBlock(c);
}

Relation MMUL_Join::join(Relation rel1, Relation rel2) {
    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));
    hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH);

    hipsparseSpMatDescr_t mat1, mat2, matOut;

    std::stringstream name;
    name << "MMUL Join (" << rel1.count << ", " << rel2.count << ")";
    Timer t(name.str().c_str());

    int alpha = 1;
    int beta = 0;

    OUT_MAT *outMatrix;
    if (rel1.count <= 0 || rel2.count <= 0) {
        Relation outRel;
        outRel.count = 0;
        hipMalloc(&outRel.data, 0);
        return outRel;
    }
    int outSize = dimA * dimC * sizeof(OUT_MAT);
    CUDA_CHECK(hipMalloc(&outMatrix, outSize));

    IN_MAT* M1 = rel1.toDenseMatrix(dimA, dimB);
    IN_MAT* M2 = rel2.toDenseMatrix(dimB, dimC);
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipGetLastError());

    t.lap("Relation to Matrix");

    hipblasStatus_t mmul_status = hipblasGemmEx(handle, 
        HIPBLAS_OP_N, HIPBLAS_OP_N,
        dimA, dimC, dimB, &alpha,
        M1, HIP_R_8I, dimA,
        M2, HIP_R_8I, dimB,
        &beta,
        outMatrix, HIP_R_32I, dimC,
        HIP_R_32I,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);

    hipDeviceSynchronize();
    CUBLAS_CHECK(mmul_status);
    CUDA_CHECK(hipGetLastError());

    t.lap("Matrix Multiplication");
    
    Relation outRel(outMatrix, dimA, dimC);
    hipDeviceSynchronize();
    CUDA_CHECK(hipGetLastError());
    
    t.lap("Matrix to Relation");

    CUDA_CHECK(hipFree(outMatrix));
    CUDA_CHECK(hipFree(M1));
    CUDA_CHECK(hipFree(M2));
    
    t.finish();
    hipblasDestroy(handle);

    return outRel;
}
